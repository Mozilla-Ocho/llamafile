#include "hip/hip_runtime.h"
#define CUB_IGNORE_DEPRECATED_CPP_DIALECT
#include "whisper.cpp/whisper-mel-cuda.hpp"
#include "whisper.h"

#include <ggml-backend.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipblas.h>
#include <hip/hip_complex.h>
#include <cub/device/device_reduce.cuh>
#include <>

#include <algorithm>

#if defined(_MSC_VER)
#pragma warning(disable: 4324) // added padding
#endif

namespace {

static const char* cufftGetErrorString(hipfftResult_t res) {
    switch (res) {
    case HIPFFT_SUCCESS: return "The cuFFT operation was successful";
    case HIPFFT_INVALID_PLAN: return "cuFFT was passed an invalid plan handle";
    case HIPFFT_ALLOC_FAILED: return "cuFFT failed to allocate GPU or CPU memory";
    case HIPFFT_INVALID_TYPE: return "No longer used";
    case HIPFFT_INVALID_VALUE: return "User specified an invalid pointer or parameter";
    case HIPFFT_INTERNAL_ERROR: return "Driver or internal cuFFT library error";
    case HIPFFT_EXEC_FAILED: return "Failed to execute an FFT on the GPU";
    case HIPFFT_SETUP_FAILED: return "The cuFFT library failed to initialize";
    case HIPFFT_INVALID_SIZE: return "User specified an invalid transform size";
    case HIPFFT_UNALIGNED_DATA: return "No longer used";
    case HIPFFT_INCOMPLETE_PARAMETER_LIST: return "Missing parameters in call";
    case HIPFFT_INVALID_DEVICE: return "Execution of a plan was on different GPU than plan creation";
    case HIPFFT_PARSE_ERROR: return "Internal plan database error";
    case HIPFFT_NO_WORKSPACE: return "No workspace has been provided prior to plan execution";
    case HIPFFT_NOT_IMPLEMENTED: return "Function does not implement functionality for parameters given.";
    case CUFFT_LICENSE_ERROR: return "Used in previous versions.";
    case HIPFFT_NOT_SUPPORTED: return "Operation is not supported for parameters given.";
    default: return "Unknown error";
    }
}

#define CUFFT_CHECK(err) CUDA_CHECK_GEN(err, HIPFFT_SUCCESS, cufftGetErrorString)

__global__ void k_fill_stft_input(
    const float * padded_samples,
    const int n_frames,
    const float * hann_window,
    float * stft_in
) {
    auto y = blockIdx.y * blockDim.y + threadIdx.y;
    // if (y >= n_frames) return;
    auto x = blockIdx.x * blockDim.x + threadIdx.x;
    // if (x >= WHISPER_N_FFT) return;

    auto line = padded_samples + y * WHISPER_HOP_LENGTH;
    auto outLine = stft_in + y * WHISPER_N_FFT;

    outLine[x] = line[x] * hann_window[x];
}

__global__ void k_calc_magnitudes(
    const hipComplex * stft_out,
    const int n_frames,
    float * magnitudes
) {
    auto y = blockIdx.y * blockDim.y + threadIdx.y;
    // if (y >= n_frames) return;
    auto x = blockIdx.x * blockDim.x + threadIdx.x;
    // if (x >= WHISPER_N_FFT_HALF) return;

    auto idx = y * WHISPER_N_FFT_HALF + x;

    auto r = stft_out[idx].x;
    auto i = stft_out[idx].y;
    magnitudes[idx] = r * r + i * i;
}

__global__ void k_calc_log_mel(
    const float * mel_data,
    const int n_mel,
    const float * max_val,
    float * log_mel
) {
    auto x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= n_mel) return;

    float val = mel_data[x];

    constexpr float e = 1e-10f;
    if (val < e) val = e;

    val = log10(val);

    const float max = log10(*max_val) - 8.f;
    if (val < max) val = max;

    log_mel[x] = (val + 4) / 4;
}

static void fill_stft_input(
    const float * padded_samples,
    int n_frames,
    const float * hann_window,
    float * stft_in,
    hipStream_t stream
) {
    dim3 block(WHISPER_N_FFT, 1);
    dim3 grid(1, n_frames);

    k_fill_stft_input<<<grid, block, 0, stream>>>(padded_samples, n_frames, hann_window, stft_in);
}

static void calc_magnitudes(
    const hipComplex * stft_out,
    int n_frames,
    float * magnitudes,
    hipStream_t stream
) {
    dim3 block(WHISPER_N_FFT_HALF, 1);
    dim3 grid(1, n_frames);
    k_calc_magnitudes<<<grid, block, 0, stream>>>(stft_out, n_frames, magnitudes);
}

constexpr auto LOG_MEL_PREFIX_SIZE = 256;

static void calc_log_mel(
    const float * mel_data,
    int n_mel,
    void * tempStorage,
    int tempStorageSize,
    float * log_mel,
    hipStream_t stream
) {
    float * max_val = reinterpret_cast<float *>(tempStorage);
    void * maxTemp = reinterpret_cast<char*>(tempStorage) + LOG_MEL_PREFIX_SIZE;

    size_t nbytes = size_t(tempStorageSize - LOG_MEL_PREFIX_SIZE);
    hipcub::DeviceReduce::Max(maxTemp, nbytes, mel_data, max_val, n_mel, stream);

    int block = 256;
    int grid = (n_mel + block - 1) / block;

    k_calc_log_mel<<<grid, block, 0, stream>>>(mel_data, n_mel, max_val, log_mel);
}

class mel_calc_cuda : public whisper_mel_calc {
    const int m_n_mel;

    ggml_backend_t m_backend = nullptr;
    int m_device = -1;

    hipStream_t m_stream = nullptr;
    hipblasHandle_t m_cublas_handle = nullptr;

    float * m_hann_window = nullptr;

    float * m_filters = nullptr;

    // max samples for which we have allocated memory for the temp working areas below (cufft, log_mel)
    int m_n_max_samples = 0;

    size_t m_cufft_workspace_size = 0;
    void * m_cufft_workspace = nullptr;

    size_t m_log_mel_temp_storage_size = 0;
    void * m_log_mel_temp_storage = nullptr;
public:
    mel_calc_cuda(ggml_backend_t backend, const whisper_filters & filters)
        : m_n_mel(filters.n_mel)
        , m_backend(backend)
    {
        ggml_backend_cuda_context* cuda_ctx = (ggml_backend_cuda_context*)m_backend->context;
        m_device = cuda_ctx->device;

        if (ggml_cuda_info().devices[m_device].cc < 600) {
            // we've only tesed on 6.0 and higher and we've had reports of crashes on 5.0:
            // https://github.com/ggerganov/whisper.cpp/issues/2230
            // to be safe forbid anything below 6.0
            throw std::runtime_error("CUDA compute capability 6.0 or higher is required");
        }

        ggml_cuda_set_device(m_device);

        if (filters.n_fft != WHISPER_N_FFT_HALF) {
            throw std::invalid_argument("MelFilters n_frames must be WHISPER_N_FFT_HALF");
        }
        assert(filters.data.size() == filters.n_mel * WHISPER_N_FFT_HALF);

        CUDA_CHECK(hipStreamCreate(&m_stream));
        CUBLAS_CHECK(hipblasCreate(&m_cublas_handle));
        CUBLAS_CHECK(hipblasSetMathMode(m_cublas_handle, HIPBLAS_TF32_TENSOR_OP_MATH));
        CUBLAS_CHECK(hipblasSetStream(m_cublas_handle, m_stream));

        // create Hann window
        {
            auto hw = whisper_mel_calc::hann_window();
            CUDA_CHECK(hipMallocAsync(&m_hann_window, hw.len * sizeof(float), m_stream));
            CUDA_CHECK(hipMemcpyAsync(m_hann_window, hw.data, hw.len * sizeof(float), hipMemcpyHostToDevice, m_stream));
        }

        // fill filters
        {
            auto& f = filters.data;
            CUDA_CHECK(hipMallocAsync(&m_filters, f.size() * sizeof(float), m_stream));
            CUDA_CHECK(hipMemcpyAsync(m_filters, f.data(), f.size() * sizeof(float), hipMemcpyHostToDevice, m_stream));
        }

        // preallocate working areas enough for the most common cases (<= 30s)
        ensure_working_areas(WHISPER_N_SAMPLES);
    }

    ~mel_calc_cuda() {
        ggml_cuda_set_device(m_device);
        CUDA_CHECK(hipStreamSynchronize(m_stream));
        CUDA_CHECK(hipStreamDestroy(m_stream));
        CUDA_CHECK(hipFree(m_hann_window));
        CUDA_CHECK(hipFree(m_cufft_workspace));
        CUDA_CHECK(hipFree(m_filters));
        CUDA_CHECK(hipFree(m_log_mel_temp_storage));
    }

    void ensure_working_areas(int n_samples) {
        if (n_samples <= m_n_max_samples) {
            return;
        }

        const auto max_padded_samples = n_samples + WHISPER_N_SAMPLES + WHISPER_N_FFT;
        const auto max_frames = 1 + (max_padded_samples - WHISPER_N_FFT) / WHISPER_HOP_LENGTH;

        // cufft workspace
        {
            if (m_cufft_workspace) {
                CUDA_CHECK(hipFree(m_cufft_workspace));
                m_cufft_workspace_size = 0;
                m_cufft_workspace = nullptr;
            }
            CUFFT_CHECK(hipfftEstimate1d(WHISPER_N_FFT, HIPFFT_R2C, max_frames, &m_cufft_workspace_size));
            CUDA_CHECK(hipMallocAsync(&m_cufft_workspace, m_cufft_workspace_size, m_stream));
        }

        // device reduce working area
        {
            if (m_log_mel_temp_storage) {
                CUDA_CHECK(hipFree(m_log_mel_temp_storage));
                m_log_mel_temp_storage_size = 0;
                m_log_mel_temp_storage = nullptr;
            }

            const auto max_mels = 160;

            size_t nbytes = 0;
            float* temp = nullptr;
            hipcub::DeviceReduce::Max(nullptr, nbytes, temp, temp, max_frames * max_mels);
            m_log_mel_temp_storage_size = nbytes + LOG_MEL_PREFIX_SIZE;

            CUDA_CHECK(hipMallocAsync(&m_log_mel_temp_storage, m_log_mel_temp_storage_size, m_stream));
        }

        m_n_max_samples = n_samples;
    }

    virtual whisper_mel calculate(whisper_span<const float> samples, int /*n_threads*/) override {
        ggml_cuda_set_device(m_device);
        ensure_working_areas(samples.len);

        const size_t mirror_pad = WHISPER_N_FFT / 2;
        const size_t padded_size = samples.len + WHISPER_N_SAMPLES + WHISPER_N_FFT;

        // pad
        std::vector<float> padded_samples(padded_size);
        std::reverse_copy(samples.data + 1, samples.data + 1 + mirror_pad, padded_samples.begin()); // reflect
        std::copy(samples.data, samples.data + samples.len, padded_samples.begin() + mirror_pad); // copy

        // fill the rest of the data
        // it should canonically be mirrored at the end as well,
        // but we just assume the last MEL_FRAME_SIZE/2 samples are zeros
        std::fill(padded_samples.begin() + mirror_pad + samples.len, padded_samples.end(), 0.f);

        const auto n_frames = 1 + (padded_samples.size() - WHISPER_N_FFT) / WHISPER_HOP_LENGTH;

        float * cu_padded_samples = nullptr;
        CUDA_CHECK(hipMallocAsync(&cu_padded_samples, padded_samples.size() * sizeof(float), m_stream));
        CUDA_CHECK(hipMemcpyAsync(cu_padded_samples, padded_samples.data(), padded_samples.size() * sizeof(float), hipMemcpyHostToDevice, m_stream));

        float * stft_in = nullptr; // contiguous buffer for stft input
        CUDA_CHECK(hipMallocAsync(&stft_in, n_frames * WHISPER_N_FFT * sizeof(float), m_stream));

        fill_stft_input(cu_padded_samples, int(n_frames), m_hann_window, stft_in, m_stream);

        hipfftComplex* stft_out;
        CUDA_CHECK(hipMallocAsync(&stft_out, n_frames * WHISPER_N_FFT_HALF * sizeof(hipfftComplex), m_stream));

        hipfftHandle plan;
        CUFFT_CHECK(hipfftCreate(&plan));
        CUFFT_CHECK(hipfftSetAutoAllocation(plan, 0));
        {
            size_t waSize;
            CUFFT_CHECK(hipfftMakePlan1d(plan, WHISPER_N_FFT, HIPFFT_R2C, int(n_frames), &waSize));
            assert(waSize <= m_cufft_workspace_size);
            CUFFT_CHECK(hipfftSetWorkArea(plan, m_cufft_workspace));
            CUFFT_CHECK(hipfftSetStream(plan, m_stream));
        }
        CUFFT_CHECK(hipfftExecR2C(plan, stft_in, stft_out));

        const auto n_mag_frames = n_frames - 1; // drop last frame
        float * magnitudes;
        CUDA_CHECK(hipMallocAsync(&magnitudes, n_mag_frames * WHISPER_N_FFT_HALF * sizeof(float), m_stream));
        calc_magnitudes(stft_out, int(n_mag_frames), magnitudes, m_stream);

        float * mel_data = nullptr;
        CUDA_CHECK(hipMallocAsync(&mel_data, m_n_mel * n_mag_frames * sizeof(float), m_stream));

        const float fone = 1.0f, fzero = 0.0f;
        CUBLAS_CHECK(hipblasSgemm(m_cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
            int(n_mag_frames), m_n_mel, WHISPER_N_FFT_HALF,
            &fone,
            magnitudes, WHISPER_N_FFT_HALF,
            m_filters, WHISPER_N_FFT_HALF,
            &fzero,
            mel_data, int(n_mag_frames)));

        whisper_mel ret;
        // Calculate semi-padded sample length to ensure compatibility
        int n_len_org = 1 + int(samples.len + mirror_pad - WHISPER_N_FFT) / WHISPER_HOP_LENGTH;
        whisper_mel_init(ret, m_backend, int(n_mag_frames), n_len_org, m_n_mel);
        assert(ggml_nbytes(ret.tensor) == m_n_mel * n_mag_frames * sizeof(float));

        float* log_mels = reinterpret_cast<float*>(ret.tensor->data);

        calc_log_mel(
            mel_data, int(m_n_mel * n_mag_frames),
            m_log_mel_temp_storage , int(m_log_mel_temp_storage_size),
            log_mels, m_stream);

        CUDA_CHECK(hipStreamSynchronize(m_stream));

        // cleanup
        CUFFT_CHECK(hipfftDestroy(plan));
        CUDA_CHECK(hipFreeAsync(mel_data, m_stream));
        CUDA_CHECK(hipFreeAsync(magnitudes, m_stream));
        CUDA_CHECK(hipFreeAsync(stft_out, m_stream));
        CUDA_CHECK(hipFreeAsync(stft_in, m_stream));
        CUDA_CHECK(hipFreeAsync(cu_padded_samples, m_stream));

        return ret;
    }
};

}

whisper_mel_calc * whisper_mel_calc_create_cuda(ggml_backend_t backend, const whisper_filters & filters) {
    try {
        return new mel_calc_cuda(backend, filters);
    }
    catch (...) {
        // TODO: log error (but for this we would have to expose the log state to be accessible here)
        return nullptr;
    }
}
