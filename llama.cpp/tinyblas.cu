#include "hip/hip_runtime.h"
#include "tinyblas.h"

#define READ(A, trans, ld, i, j) \
  (((trans) == HIPBLAS_OP_N) ? (A)[(i) + (j) * (ld)] : (A)[(j) + (i) * (ld)])
#define READ16(...) __half2float(READ(__VA_ARGS__))

static __device__ __forceinline__ void matmul(int m, int n, int k,
                                              const half *A, int lda,
                                              const half *B, int ldb,
                                              half       *C, int ldc) {
  for (int i = 0; i < m; ++i) {
    for (int j = 0; j < n; ++j) {
      float sum = 0.0;
      half *cptr = C + i + j * ldc;
      for (int l = 0; l < k; ++l) {
        sum += READ16(A, HIPBLAS_OP_T, lda, i, l) *
               READ16(B, HIPBLAS_OP_N, ldb, l, j);
      }
      *cptr = __float2half(sum);
    }
  }
}

static __global__ void wrap_matmul(int m, int n, int k, const half *A, int lda,
                                   const half *B, int ldb, half *C, int ldc) {
  matmul(m, n, k, A, lda, B, ldb, C, ldc);
}

static __global__ void matmul32(int m, int n, int k, const float *A, int lda,
                                const float *B, int ldb, float *C, int ldc) {
  for (int i = 0; i < m; ++i) {
    for (int j = 0; j < n; ++j) {
      float sum = 0.0;
      float *cptr = C + i + j * ldc;
      for (int l = 0; l < k; ++l) {
        sum += READ(A, HIPBLAS_OP_T, lda, i, l) *
               READ(B, HIPBLAS_OP_N, ldb, l, j);
      }
      *cptr = sum;
    }
  }
}

static bool check_args(hipblasOperation_t transa, hipblasOperation_t transb,
                       const void *pAlpha, hipDataType Atype,
                       hipDataType Btype, const void *pBeta,
                       hipDataType Ctype, hipblasComputeType_t computeType) {
  return transa == HIPBLAS_OP_T && transb == HIPBLAS_OP_N &&
    Atype == HIP_R_16F && Btype == HIP_R_16F && Ctype == HIP_R_16F &&
    computeType == HIPBLAS_COMPUTE_16F &&
    __half2float(*(half *)pAlpha) == 1.0f &&
    __half2float(*(half *)pBeta) == 0.0f;
}

hipblasStatus_t tinyblasSgemm(hipStream_t stream,
                             hipblasOperation_t transa,
                             hipblasOperation_t transb,
                             int m, int n, int k,
                             const float           *alpha,
                             const float           *A, int lda,
                             const float           *B, int ldb,
                             const float           *beta,
                             float           *C, int ldc) {
  if (transa != HIPBLAS_OP_T || transb != HIPBLAS_OP_N ||
      *alpha != 1.0f || *beta != 0.0f) {
    return HIPBLAS_STATUS_NOT_SUPPORTED;
  }
  matmul32<<<1, 1, 0, stream>>>(m, n, k, A, lda, B, ldb, C, ldc);
  return HIPBLAS_STATUS_SUCCESS;
}

// https://docs.nvidia.com/cuda/cublas/index.html#cublasgemmex

hipblasStatus_t tinyblasGemmEx(hipStream_t stream,
                              hipblasOperation_t transa,
                              hipblasOperation_t transb,
                              int m,
                              int n,
                              int k,
                              const void    *alpha,
                              const void     *A,
                              hipDataType Atype,
                              int lda,
                              const void     *B,
                              hipDataType Btype,
                              int ldb,
                              const void    *beta,
                              void           *C,
                              hipDataType Ctype,
                              int ldc,
                              hipblasComputeType_t computeType,
                              hipblasGemmAlgo_t algo) {
  if (!check_args(transa, transb, alpha, Atype, Btype, beta, Ctype,
                  computeType)) {
    return HIPBLAS_STATUS_NOT_SUPPORTED;
  }

  wrap_matmul<<<1, 1, 0, stream>>>(
      m, n, k, (const half*)A, lda, (const half *)B, ldb, (half *)C, ldc);
  return HIPBLAS_STATUS_SUCCESS;
}

// https://docs.nvidia.com/cuda/cublas/index.html#cublasgemmbatchedex

static __global__ void tinyblasGBE_entry(int m, int n, int k,
                                         const half *const  Aarray[],
                                         int lda,
                                         const half *const  Barray[],
                                         int ldb,
                                         half *const        Carray[],
                                         int ldc,
                                         int batchCount) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int jump = blockDim.x * gridDim.x;

  for (; x < batchCount; x += jump) {
    matmul(m, n, k, Aarray[x], lda, Barray[x], ldb, Carray[x], ldc);
  }
}

hipblasStatus_t tinyblasGemmBatchedEx(hipStream_t stream,
                                     hipblasOperation_t transa,
                                     hipblasOperation_t transb,
                                     int m,
                                     int n,
                                     int k,
                                     const void    *alpha,
                                     const void     *const Aarray[],
                                     hipDataType Atype,
                                     int lda,
                                     const void     *const Barray[],
                                     hipDataType Btype,
                                     int ldb,
                                     const void    *beta,
                                     void           *const Carray[],
                                     hipDataType Ctype,
                                     int ldc,
                                     int batchCount,
                                     hipblasComputeType_t computeType,
                                     hipblasGemmAlgo_t algo) {
  if (!check_args(transa, transb, alpha, Atype, Btype, beta, Ctype,
                  computeType)) {
    return HIPBLAS_STATUS_NOT_SUPPORTED;
  }

  // https://developer.nvidia.com/blog/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
  int numSMs, devId;
  hipGetDevice(&devId);
  hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, devId);
  int maxblocks = 16 * numSMs;
  int maxthreads = 128;

  tinyblasGBE_entry<<<maxblocks, maxthreads, 0, stream>>>(
      m, n, k, (const half **)Aarray, lda, (const half **)Barray, ldb,
      (half **)Carray, ldc, batchCount);
  return HIPBLAS_STATUS_SUCCESS;
}

// https://docs.nvidia.com/cuda/cublas/index.html#cublasgemmstridedbatchedex

#define STRIDE0(A, i, stride) ((A) + (i) * (stride))
#define STRIDE(A, type, i, stride)                            \
  ((type) == HIP_R_16F                                       \
   ? (void *)STRIDE0((half *)(A), (i), (stride))              \
   : (void *)STRIDE0((float *)(A), (i), (stride)))
#define STRIDE_CONST(A, type, i, stride)                      \
  ((type) == HIP_R_16F                                       \
   ? (const void *)STRIDE0((const half *)(A), (i), (stride))  \
   : (const void *)STRIDE0((const float *)(A), (i), (stride)))

static __global__ void tinyblasGSBE_entry(int m, int n, int k,
                                          const half      *A,
                                          int             lda,
                                          long long int   strideA,
                                          const half      *B,
                                          int             ldb,
                                          long long int   strideB,
                                          half            *C,
                                          int             ldc,
                                          long long int   strideC,
                                          int batchCount) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int jump = blockDim.x * gridDim.x;

  for (; x < batchCount; x += jump) {
    matmul(m, n, k, A + x * strideA, lda, B + x * strideB, ldb, C + x * strideC,
           ldc);
  }
}

hipblasStatus_t tinyblasGemmStridedBatchedEx(hipStream_t stream,
                                            hipblasOperation_t transa,
                                            hipblasOperation_t transb,
                                            int m, int n, int k,
                                            const void    *pAlpha,
                                            const void     *A,
                                            hipDataType Atype,
                                            int lda,
                                            long long int strideA,
                                            const void     *B,
                                            hipDataType Btype,
                                            int ldb,
                                            long long int strideB,
                                            const void    *pBeta,
                                            void           *C,
                                            hipDataType Ctype,
                                            int ldc,
                                            long long int strideC,
                                            int batchCount,
                                            hipblasComputeType_t computeType,
                                            hipblasGemmAlgo_t algo) {
  if (!check_args(transa, transb, pAlpha, Atype, Btype, pBeta, Ctype,
                  computeType)) {
    return HIPBLAS_STATUS_NOT_SUPPORTED;
  }

  // https://developer.nvidia.com/blog/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
  int numSMs, devId;
  hipGetDevice(&devId);
  hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, devId);
  int maxblocks = 16 * numSMs;
  int maxthreads = 128;

  // call the entry function
  tinyblasGSBE_entry<<<maxblocks, maxthreads, 0, stream>>>(
      m, n, k, (const half*)A, lda, strideA, (const half*)B, ldb, strideB,
      (half *)C, ldc, strideC, batchCount);

  return HIPBLAS_STATUS_SUCCESS;
}
