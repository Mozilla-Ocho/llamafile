#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>

#define READ(A, trans, ld, i, j) \
  __half2float(((trans) == HIPBLAS_OP_N) \
                 ? (A)[(i) + (j) * (ld)] : (A)[(j) + (i) * (ld)])

static __device__ __forceinline__ void matmul(int m, int n, int k,
                                              const half *A, int lda,
                                              const half *B, int ldb,
                                              half       *C, int ldc) {
  for (int i = 0; i < m; ++i) {
    for (int j = 0; j < n; ++j) {
      float sum = 0.0;
      half *cptr = C + i + j * ldc;
      for (int l = 0; l < k; ++l) {
        sum += READ(A, HIPBLAS_OP_T, lda, i, l) *
               READ(B, HIPBLAS_OP_N, ldb, l, j);
      }
      *cptr = __float2half(sum);
    }
  }
}

static __global__ void wrap_matmul(int m, int n, int k, const half *A, int lda,
                                   const half *B, int ldb, half *C, int ldc) {
  matmul(m, n, k, A, lda, B, ldb, C, ldc);
}

static bool check_args(hipblasOperation_t transa, hipblasOperation_t transb,
                       const void *pAlpha, hipDataType Atype,
                       hipDataType Btype, const void *pBeta,
                       hipDataType Ctype, hipblasComputeType_t computeType) {
  return transa == HIPBLAS_OP_T && transb == HIPBLAS_OP_N &&
    Atype == HIP_R_16F && Btype == HIP_R_16F && Ctype == HIP_R_16F &&
    computeType == HIPBLAS_COMPUTE_16F &&
    __half2float(*(half *)pAlpha) == 1.0f &&
    __half2float(*(half *)pBeta) == 0.0f;
}

// https://docs.nvidia.com/cuda/cublas/index.html#cublasgemmex

hipblasStatus_t hipblasGemmEx(hipblasHandle_t handle,
                            hipblasOperation_t transa,
                            hipblasOperation_t transb,
                            int m,
                            int n,
                            int k,
                            const void    *alpha,
                            const void     *A,
                            hipDataType Atype,
                            int lda,
                            const void     *B,
                            hipDataType Btype,
                            int ldb,
                            const void    *beta,
                            void           *C,
                            hipDataType Ctype,
                            int ldc,
                            hipblasComputeType_t computeType,
                            hipblasGemmAlgo_t algo) {
  if (!check_args(transa, transb, alpha, Atype, Btype, beta, Ctype,
                  computeType)) {
    return HIPBLAS_STATUS_NOT_SUPPORTED;
  }

  hipStream_t stream;
  hipblasGetStream(handle, &stream);
  wrap_matmul<<<1, 1, 0, stream>>>(
      m, n, k, (const half*)A, lda, (const half *)B, ldb, (half *)C, ldc);
  return HIPBLAS_STATUS_SUCCESS;
}

// https://docs.nvidia.com/cuda/cublas/index.html#cublasgemmstridedbatchedex

#define STRIDE0(A, i, stride) ((A) + (i) * (stride))
#define STRIDE(A, type, i, stride)                            \
  ((type) == HIP_R_16F                                       \
   ? (void *)STRIDE0((half *)(A), (i), (stride))              \
   : (void *)STRIDE0((float *)(A), (i), (stride)))
#define STRIDE_CONST(A, type, i, stride)                      \
  ((type) == HIP_R_16F                                       \
   ? (const void *)STRIDE0((const half *)(A), (i), (stride))  \
   : (const void *)STRIDE0((const float *)(A), (i), (stride)))

static __global__ void cublasGSBE_entry(int m, int n, int k,
                                        const half      *A,
                                        int             lda,
                                        long long int   strideA,
                                        const half      *B,
                                        int             ldb,
                                        long long int   strideB,
                                        half            *C,
                                        int             ldc,
                                        long long int   strideC,
                                        int batchCount) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int jump = blockDim.x * gridDim.x;

  for (; x < batchCount; x += jump) {
    matmul(m, n, k, A + x * strideA, lda, B + x * strideB, ldb, C + x * strideC,
           ldc);
  }
}

hipblasStatus_t hipblasGemmStridedBatchedEx(hipblasHandle_t handle,
                                          hipblasOperation_t transa,
                                          hipblasOperation_t transb,
                                          int m, int n, int k,
                                          const void    *pAlpha,
                                          const void     *A,
                                          hipDataType Atype,
                                          int lda,
                                          long long int strideA,
                                          const void     *B,
                                          hipDataType Btype,
                                          int ldb,
                                          long long int strideB,
                                          const void    *pBeta,
                                          void           *C,
                                          hipDataType Ctype,
                                          int ldc,
                                          long long int strideC,
                                          int batchCount,
                                          hipblasComputeType_t computeType,
                                          hipblasGemmAlgo_t algo) {
  if (!check_args(transa, transb, pAlpha, Atype, Btype, pBeta, Ctype,
                  computeType)) {
    return HIPBLAS_STATUS_NOT_SUPPORTED;
  }

  hipStream_t stream;
  hipblasGetStream(handle, &stream);

  // https://developer.nvidia.com/blog/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
  int numSMs, devId;
  hipGetDevice(&devId);
  hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, devId);
  int maxblocks = 16 * numSMs;
  int maxthreads = 128;

  // call the entry function
  cublasGSBE_entry<<<maxblocks, maxthreads, 0, stream>>>(
      m, n, k, (const half*)A, lda, strideA, (const half*)B, ldb, strideB,
      (half *)C, ldc, strideC, batchCount);

  return HIPBLAS_STATUS_SUCCESS;
}
