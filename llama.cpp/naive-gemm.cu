#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>

#define MULZERO(X, Y) (fabs((X)) > 0 ? (X) * (Y) : 0.0)
#define READ0(A, trans, ld, i, j) \
  (((trans) == HIPBLAS_OP_N) ? (A)[(i) + (j) * (ld)] : (A)[(j) + (i) * (ld)])
#define READ(A, type, trans, ld, i, j) \
  ((type) == HIP_R_16F                                         \
   ? __half2float(READ0((half *)(A), (trans), (ld), (i), (j)))  \
   : READ0((float *)(A), (trans), (ld), (i), (j)))

static __device__ __forceinline__ void matmul(hipblasOperation_t transa,
                                              hipblasOperation_t transb,
                                              int m, int n, int k,
                                              float               alpha,
                                              const void          *A,
                                              hipDataType    Atype,
                                              int lda,
                                              const void          *B,
                                              hipDataType    Btype,
                                              int ldb,
                                              float               beta,
                                              void                *C,
                                              hipDataType    Ctype,
                                              int ldc) {
  for (int i = 0; i < m; ++i) {
    for (int j = 0; j < n; ++j) {
      float sum = 0.0;
      for (int l = 0; l < k; ++l) {
        sum += READ(A, Atype, transa, lda, i, l) *
               READ(B, Btype, transb, ldb, l, j);
      }
      if (Ctype == HIP_R_16F) {
        half *cptr = (half *)C + i + ldc * j;
        *cptr = __float2half(MULZERO(alpha, sum) +
                             MULZERO(beta, __half2float(*cptr)));
      } else {
        float *cptr = (float *)C + i + ldc * j;
        *cptr = MULZERO(alpha, sum) + MULZERO(beta, *cptr);
      }
    }
  }
}

// https://docs.nvidia.com/cuda/cublas/index.html#cublasgemmstridedbatchedex

#define STRIDE0(A, i, stride) ((A) + (i) * (stride))
#define STRIDE(A, type, i, stride)                            \
  ((type) == HIP_R_16F                                       \
   ? (void *)STRIDE0((half *)(A), (i), (stride))              \
   : (void *)STRIDE0((float *)(A), (i), (stride)))
#define STRIDE_CONST(A, type, i, stride)                      \
  ((type) == HIP_R_16F                                       \
   ? (const void *)STRIDE0((const half *)(A), (i), (stride))  \
   : (const void *)STRIDE0((const float *)(A), (i), (stride)))

static __global__ void cublasGSBE_entry(hipblasOperation_t transa,
                                        hipblasOperation_t transb,
                                        int m, int n, int k,
                                        float           alpha,
                                        const void      *A,
                                        hipDataType  Atype,
                                        int             lda,
                                        long long int   strideA,
                                        const void      *B,
                                        hipDataType  Btype,
                                        int             ldb,
                                        long long int   strideB,
                                        float           beta,
                                        void            *C,
                                        hipDataType  Ctype,
                                        int             ldc,
                                        long long int   strideC,
                                        int batchCount) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int jump = blockDim.x * gridDim.x;

  const void *A_x;
  const void *B_x;
  void *C_x;

  for (; x < batchCount; x += jump) {
    A_x = STRIDE_CONST(A, Atype, x, strideA);
    B_x = STRIDE_CONST(B, Btype, x, strideB);
    C_x = STRIDE(C, Ctype, x, strideC);
    matmul(transa, transb, m, n, k, alpha, A_x, Atype, lda, B_x, Btype, ldb,
           beta, C_x, Ctype, ldc);
  }
}

hipblasStatus_t hipblasGemmStridedBatchedEx(hipblasHandle_t handle,
                                          hipblasOperation_t transa,
                                          hipblasOperation_t transb,
                                          int m, int n, int k,
                                          const void    *pAlpha,
                                          const void     *A,
                                          hipDataType Atype,
                                          int lda,
                                          long long int strideA,
                                          const void     *B,
                                          hipDataType Btype,
                                          int ldb,
                                          long long int strideB,
                                          const void    *pBeta,
                                          void           *C,
                                          hipDataType Ctype,
                                          int ldc,
                                          long long int strideC,
                                          int batchCount,
                                          hipblasComputeType_t computeType,
                                          hipblasGemmAlgo_t algo) {
  if ((Atype != HIP_R_16F && Atype != HIP_R_32F) ||
      (Btype != HIP_R_16F && Btype != HIP_R_32F) ||
      (Ctype != HIP_R_16F && Ctype != HIP_R_32F) ||
      (transa != HIPBLAS_OP_N && transa != HIPBLAS_OP_T) ||
      (transb != HIPBLAS_OP_N && transb != HIPBLAS_OP_T)) {
    return HIPBLAS_STATUS_NOT_SUPPORTED;
  }

  float alpha, beta;
  switch (computeType) {
    case HIPBLAS_COMPUTE_16F:
      beta = __half2float(*(half *)pBeta);
      alpha = __half2float(*(half *)pAlpha);
      break;
    case HIPBLAS_COMPUTE_32F:
      beta = *(float *)pBeta;
      alpha = *(float *)pAlpha;
      break;
    default:
      return HIPBLAS_STATUS_NOT_SUPPORTED;
  }

  // https://developer.nvidia.com/blog/cuda-pro-tip-write-flexible-kernels-grid-stride-loops/
  int numSMs, devId;
  hipGetDevice(&devId);
  hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, devId);
  int maxblocks = 16 * numSMs;
  int maxthreads = 128;

  // call the entry function
  cublasGSBE_entry<<<maxblocks, maxthreads>>>(transa, transb, m, n, k, alpha,
                                              A, Atype, lda, strideA, B, Btype,
                                              ldb, strideB, beta, C, Ctype,
                                              ldc, strideC, batchCount);

  return HIPBLAS_STATUS_SUCCESS;
}
