#include "hip/hip_runtime.h"
// -*- mode:c;indent-tabs-mode:nil;c-basic-offset:4;coding:utf-8 -*-
// vi: set et ft=c ts=4 sts=4 sw=4 fenc=utf-8 :vi
//
// Copyright 2023 Mozilla Foundation
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "tinyblas.h"

#define READ(A, trans, ld, i, j)                                        \
    (((trans) == TINYBLAS_OP_N) ? (A)[(i) + (j) * (ld)] : (A)[(j) + (i) * (ld)])
#define READ16(A, trans, ld, i, j) __half2float(READ(A, trans, ld, i, j))

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

template<int BM, int BN, int BK, int TM, int TN>
static __device__ void matmul32_block2d(int m, int n, int k, int x, int y,
                                        const float *A, int lda, float *As,
                                        const float *B, int ldb, float *Bs,
                                        void *C, int ldc) {
    const int ii0 = threadIdx.x / (BN / TN); /* {0, ..., (BM/TM) - 1} */
    const int ii1 = threadIdx.x % (BN / TN); /* {0, ..., (BN/TN) - 1} */

    float Cs[TM * TN];
    float At[TM];
    float Bt[TN];
    int i, h, j, l, blob;
    // within each block
    // we first zero out Cs
    for (j = 0; j < TM * TN; ++j) Cs[j] = 0;

    i = threadIdx.x;
    for (blob = 0; blob < k; blob += BK) {
        for (i = threadIdx.x; i < BK; i += blockDim.x) {
            for (j = 0; j < BM; ++j) As[(j * BK) + i] = 0;
            if ((blob + i) < k) {
                // we copy into As from A
                for (j = 0; j < BM && x + j < m; ++j) {
                    As[(j * BK) + i] =
                        READ(A, TINYBLAS_OP_T, lda, x + j, blob + i);
                }
            }
        }
        __syncthreads();
        
        for (i = threadIdx.x; i < BK; i += blockDim.x) {
            for (j = 0; j < BN; ++j) Bs[(i * BN) + j] = 0;
            if ((blob + i) < k) {
                // we copy into Bs from B
                for (j = 0; j < BN && y + j < n; ++j) {
                    Bs[(i * BN) + j] =
                        READ(B, TINYBLAS_OP_N, ldb, blob + i, y + j);
                }
            }
        }
        __syncthreads();


        // We matmul the blobs, basically Cs += matmul(As, Bs)
        for (l = 0; l < BK; ++l) {
            for (j = 0; j < TM; ++j) At[j] = As[(ii0 * TM + j) * BK + l];
            for (h = 0; h < TN; ++h) Bt[h] = Bs[(l * BN) + ii1 * TN + h];
            for (j = 0; j < TM; ++j) {
                for (h = 0; h < TN; ++h) {
                    Cs[j * TN + h] += At[j] * Bt[h];
                }
            }
        }
        __syncthreads();
    }
    __syncthreads();

    // We write Cs out into C
    x += ii0 * TM;
    y += ii1 * TN;
    for (j = 0; j < TM && x + j < m; ++j) {
        for (l = 0; l < TN && y + l < n; ++l) {
            *((float *)C + (x + j) + (y + l) * ldc) = Cs[j * TN + l];
        }
    }
    __syncthreads();
}

template<int BM, int BN, int BK, int TM, int TN>
static __global__ void tinyblasS_entry(int m, int n, int k,
                                       const float *A, int lda,
                                       const float *B, int ldb,
                                       float       *C, int ldc) {
    assert(blockDim.x == BK);
    int x = blockIdx.x * BM;
    const int jump1 = gridDim.x * BM;
    int y = blockIdx.y * BN;
    const int jump2 = gridDim.y * BN;

    extern __shared__ float svals[];  // shared across all threads in a block
    float *As = svals;
    float *Bs = svals + BM * BK;

    // each block handles a sub-matrix of C, of size BM * BN
    // each thread handles a sub-matrix of size TM * TN
    for (x = blockIdx.x * BM; x < m; x += jump1) {
        for (y = blockIdx.y * BN; y < n; y += jump2) {
            matmul32_block2d<BM, BN, BK, TM, TN>(m, n, k, x, y,  //
                             A, lda, As,     //
                             B, ldb, Bs,     //
                             C, ldc);
        }
    }
}

static bool check_args(tinyblasOperation_t transa, tinyblasOperation_t transb,
                       const void *pAlpha, hipDataType Atype,
                       hipDataType Btype, const void *pBeta,
                       hipDataType Ctype, tinyblasComputeType_t computeType) {
    return (transa == TINYBLAS_OP_T &&
            transb == TINYBLAS_OP_N &&
            Atype == HIP_R_16F &&
            Btype == HIP_R_16F &&
            (Ctype == HIP_R_16F ||
             Ctype == HIP_R_32F) &&
            ((computeType == TINYBLAS_COMPUTE_16F &&
              __half2float(*(half *)pAlpha) == 1.0f &&
              __half2float(*(half *)pBeta) == 0.0f) ||
             (computeType == TINYBLAS_COMPUTE_32F &&
              *(float *)pAlpha == 1.0f &&
              *(float *)pBeta == 0.0f)));
}

template <int BM, int BN, int BK, int TM, int TN>
static void tinyblasS_wrapper(tinyblasHandle_t stream, int m, int n, int k,
                              const float *A, int lda, const float *B, int ldb,
                              float *C, int ldc) {
    static_assert(BN <= BM, "threads can't read columns properly");
    static_assert((BM % TM == 0) && (BN % TN == 0),
                  "can't divide work for threads");
    static_assert(BK == ((BM * BN) / (TM * TN)),
                  "threads can't load memory properly");
    static_assert((BM * BN) <= (BM * BK) + (BK * BN),
                  "didn't allocate enough shared mem for threads");
    dim3 maxblocks(CEIL_DIV(m, BM), CEIL_DIV(n, BN), 1);
    int maxthreads = ((BM * BN) / (TM * TN));

    tinyblasS_entry<BM, BN, BK, TM, TN>
        <<<maxblocks, maxthreads, (sizeof(float) * (BM * BK + BK * BN)),
           stream>>>(m, n, k, A, lda, B, ldb, C, ldc);
}

tinyblasStatus_t tinyblasSgemm(tinyblasHandle_t stream,
                               tinyblasOperation_t transa,
                               tinyblasOperation_t transb,
                               int m, int n, int k,
                               const float *alpha,
                               const float *A, int lda,
                               const float *B, int ldb,
                               const float *beta,
                               float       *C, int ldc) {
    if (transa != TINYBLAS_OP_T || transb != TINYBLAS_OP_N ||
        *alpha != 1.0f || *beta != 0.0f) {
        return TINYBLAS_STATUS_NOT_SUPPORTED;
    }

    tinyblasS_wrapper<48, 24, 64, 6, 3>(stream, m, n, k, A, lda, B, ldb, C, ldc);
    return TINYBLAS_STATUS_SUCCESS;
}

template<int BM, int BN, int BK>
static __device__ void matmul_block2d(int m, int n, int k, int x, int y,
                                      const half *A, int lda, float *As,
                                      const half *B, int ldb, float *Bs,
                                      void *C, hipDataType Ctype, int ldc,
                                      float *Cs) {
    assert(blockDim.x == BK);
    static_assert(BK == BM, "");
    static_assert(BN <= BM, "");
    const int i = threadIdx.x;
    int j, l, blob;
    // within each block
    // we first zero out Cs
    for (j = 0; j < BN; ++j) Cs[j] = 0;

    for (blob = 0; blob < k; blob += BK) {
        if (i < BK) {
            if ((blob + i) < k) {
                // we copy into As from A
                for (j = 0; j < BM && x + j < m; ++j) {
                    As[(j * BK) + i] =
                        READ16(A, TINYBLAS_OP_T, lda, x + j, blob + i);
                }
                for (; j < BM; ++j) As[(j * BK) + i] = 0;
                // we copy into Bs from B
                for (j = 0; j < BN && y + j < n; ++j) {
                    Bs[(i * BN) + j] =
                        READ16(B, TINYBLAS_OP_N, ldb, blob + i, y + j);
                }
                for (; j < BN; ++j) Bs[(i * BN) + j] = 0;
            } else {  // UNLIKELY
                for (j = 0; j < BM; ++j) As[(j * BK) + i] = 0;
                for (j = 0; j < BN; ++j) Bs[(i * BN) + j] = 0;
            }
        }
        __syncthreads();

        // We matmul the blobs, basically Cs += matmul(As, Bs)
        for (j = 0; j < BN; ++j) {
            for (l = 0; l < BK; ++l) {
                Cs[j] += As[(i * BK) + l] * Bs[(l * BN) + j];
            }
        }
        __syncthreads();
    }

    for (j = 0; j < BN;  ++j) {
        As[(i*BN) + j] = Cs[j];
    }

    // We write Cs out into C
    if (y + i < n && i < BN) {
        if (Ctype == HIP_R_16F) {
            for (j = 0; j < BM && x + j < m; ++j) {
                *((half *)C + (x + j) + (y + i) * ldc) = __float2half(As[j*BN + i]);
            }
        } else {
            for (j = 0; j < BM && x + j < m; ++j) {
                *((float *)C + (x + j) + (y + i) * ldc) = As[j*BN + i];
            }
        }
    }
    __syncthreads();
}

// https://docs.nvidia.com/cuda/cublas/index.html#cublasgemmex
template<int BM, int BN, int BK>
static __global__ void tinyblasGE_entry(int m, int n, int k, const half *A,
                                        int lda, const half *B, int ldb,
                                        void *C, hipDataType Ctype,
                                        int ldc) {
    int x = blockIdx.x * BM;
    const int jump1 = gridDim.x * BM;
    int y = blockIdx.y * BN;
    const int jump2 = gridDim.y * BN;

    extern __shared__ float svals[];  // shared across all threads in a block
    float *As = svals;
    float *Bs = svals + BM * BK;
    float Cs[BN];  // only within a particular thread

    // each block handles a sub-matrix of C, of size BM * BN
    // each thread handles a sub-row of size BN
    for (x = blockIdx.x * BM; x < m; x += jump1) {
        for (y = blockIdx.y * BN; y < n; y += jump2) {
            matmul_block2d<BM, BN, BK>(m, n, k, x, y,  //
                                       A, lda, As,     //
                                       B, ldb, Bs,     //
                                       C, Ctype, ldc, Cs);
        }
    }
}

template <int BM, int BN, int BK>
static void tinyblasGE_wrapper(tinyblasHandle_t stream, int m, int n, int k,
                               const half *A, int lda, const half *B, int ldb,
                               void *C, hipDataType Ctype, int ldc) {
    dim3 maxblocks(CEIL_DIV(m, BM), CEIL_DIV(n, BN), 1);
    int maxthreads = BK;

    tinyblasGE_entry<BM, BN, BK>
        <<<maxblocks, maxthreads, (sizeof(float) * (BM * BK + BK * BN)),
           stream>>>(m, n, k, A, lda, B, ldb, C, Ctype, ldc);
}

tinyblasStatus_t tinyblasGemmEx(tinyblasHandle_t stream,
                                tinyblasOperation_t transa,
                                tinyblasOperation_t transb,
                                int m,
                                int n,
                                int k,
                                const void    *alpha,
                                const void     *A,
                                hipDataType Atype,
                                int lda,
                                const void     *B,
                                hipDataType Btype,
                                int ldb,
                                const void    *beta,
                                void           *C,
                                hipDataType Ctype,
                                int ldc,
                                tinyblasComputeType_t computeType,
                                tinyblasGemmAlgo_t algo) {
    if (!check_args(transa, transb, alpha, Atype, Btype, beta, Ctype,
                    computeType)) {
        return TINYBLAS_STATUS_NOT_SUPPORTED;
    }

    tinyblasGE_wrapper<48, 12, 48>(stream, m, n, k, (const half *)A, lda,
                                   (const half *)B, ldb, C, Ctype, ldc);
    return TINYBLAS_STATUS_SUCCESS;
}

// https://docs.nvidia.com/cuda/cublas/index.html#cublasgemmbatchedex

template<int BM, int BN, int BK>
static __global__ void tinyblasGBE_entry(int m, int n, int k,
                                         const half *const Aarray[], int lda,
                                         const half *const Barray[], int ldb,
                                         void *const Carray[],
                                         hipDataType Ctype, int ldc,
                                         int batchCount) {
    int x = blockIdx.x * BM;
    const int jump1 = gridDim.x * BM;
    int y = blockIdx.y * BN;
    const int jump2 = gridDim.y * BN;
    int z = blockIdx.z;
    const int jump3 = gridDim.z;

    extern __shared__ float svals[];  // shared across all threads in a block
    float *As = svals;
    float *Bs = svals + BM * BK;
    float Cs[BN];  // only within a particular thread

    // each block handles a sub-matrix of C, of size BM * BN
    // each thread handles a sub-row of size BN
    for (z = blockIdx.z; z < batchCount; z += jump3) {
        for (x = blockIdx.x * BM; x < m; x += jump1) {
            for (y = blockIdx.y * BN; y < n; y += jump2) {
                matmul_block2d<BM, BN, BK>(m, n, k, x, y,       //
                                           Aarray[z], lda, As,  //
                                           Barray[z], ldb, Bs,  //
                                           Carray[z], Ctype, ldc, Cs);
            }
        }
    }
}

template<int BM, int BN, int BK>
static void tinyblasGBE_wrapper(tinyblasHandle_t stream, int m, int n, int k,
                                const half *const Aarray[], int lda,
                                const half *const Barray[], int ldb,
                                void *const Carray[], hipDataType Ctype,
                                int ldc, int batchCount) {
    dim3 maxblocks(CEIL_DIV(m, BM), CEIL_DIV(n, BN), 32);
    int maxthreads = BK;

    tinyblasGBE_entry<BM, BN, BK>
        <<<maxblocks, maxthreads, (sizeof(float) * (BM * BK + BK * BN)),
           stream>>>(m, n, k, Aarray, lda, Barray,
                     ldb, Carray, Ctype, ldc, batchCount);
}

tinyblasStatus_t tinyblasGemmBatchedEx(tinyblasHandle_t stream,
                                       tinyblasOperation_t transa,
                                       tinyblasOperation_t transb,
                                       int m,
                                       int n,
                                       int k,
                                       const void    *alpha,
                                       const void     *const Aarray[],
                                       hipDataType Atype,
                                       int lda,
                                       const void     *const Barray[],
                                       hipDataType Btype,
                                       int ldb,
                                       const void    *beta,
                                       void           *const Carray[],
                                       hipDataType Ctype,
                                       int ldc,
                                       int batchCount,
                                       tinyblasComputeType_t computeType,
                                       tinyblasGemmAlgo_t algo) {
    if (!check_args(transa, transb, alpha, Atype, Btype, beta, Ctype,
                    computeType)) {
        return TINYBLAS_STATUS_NOT_SUPPORTED;
    }

    tinyblasGBE_wrapper<48, 12, 48>(stream, m, n, k, (const half **)Aarray, lda,
                                    (const half **)Barray, ldb, Carray, Ctype,
                                    ldc, batchCount);
    return TINYBLAS_STATUS_SUCCESS;
}

// https://docs.nvidia.com/cuda/cublas/index.html#cublasgemmstridedbatchedex
template<int BM, int BN, int BK>
static __global__ void tinyblasGSBE_entry(int m, int n, int k,
                                          const half      *A,
                                          int             lda,
                                          long long int   strideA,
                                          const half      *B,
                                          int             ldb,
                                          long long int   strideB,
                                          void            *C,
                                          hipDataType  Ctype,
                                          int             ldc,
                                          long long int   strideC,
                                          int batchCount) {
    int x = blockIdx.x * BM;
    const int jump1 = gridDim.x * BM;
    int y = blockIdx.y * BN;
    const int jump2 = gridDim.y * BN;
    int z = blockIdx.z;
    const int jump3 = gridDim.z;

    extern __shared__ float svals[];  // shared across all threads in a block
    float *As = svals;
    float *Bs = svals + BM * BK;
    float Cs[BN];  // only within a particular thread

    // each block handles a sub-matrix of C, of size BM * BN
    // each thread handles a sub-row of size BN
    for (z = blockIdx.z; z < batchCount; z += jump3) {
        for (x = blockIdx.x * BM; x < m; x += jump1) {
            for (y = blockIdx.y * BN; y < n; y += jump2) {
                matmul_block2d<BM, BN, BK>(
                    m, n, k, x, y,             //
                    A + z * strideA, lda, As,  //
                    B + z * strideB, ldb, Bs,  //
                    (Ctype == HIP_R_16F ? (void *)((half *)C + z * strideC)
                                         : (void *)((float *)C + z * strideC)),
                    Ctype, ldc, Cs);
            }
        }
    }
}

template <int BM, int BN, int BK>
static void tinyblasGSBE_wrapper(tinyblasHandle_t stream, int m, int n, int k,
                                 const half *A, int lda, long long int strideA,
                                 const half *B, int ldb, long long int strideB,
                                 void *C, hipDataType Ctype, int ldc,
                                 long long int strideC, int batchCount) {
    // call the entry function
    dim3 maxblocks(CEIL_DIV(m, BM), CEIL_DIV(n, BN), 32);
    int maxthreads = BK;

    tinyblasGSBE_entry<BM, BN, BK>
        <<<maxblocks, maxthreads, (sizeof(float) * (BM * BK + BK * BN)),
           stream>>>(m, n, k,                 //
                     A, lda, strideA,         //
                     B, ldb, strideB,         //
                     C, Ctype, ldc, strideC,  //
                     batchCount);
}

tinyblasStatus_t tinyblasGemmStridedBatchedEx(tinyblasHandle_t stream,
                                              tinyblasOperation_t transa,
                                              tinyblasOperation_t transb,
                                              int m, int n, int k,
                                              const void    *pAlpha,
                                              const void     *A,
                                              hipDataType Atype,
                                              int lda,
                                              long long int strideA,
                                              const void     *B,
                                              hipDataType Btype,
                                              int ldb,
                                              long long int strideB,
                                              const void    *pBeta,
                                              void           *C,
                                              hipDataType Ctype,
                                              int ldc,
                                              long long int strideC,
                                              int batchCount,
                                              tinyblasComputeType_t computeType,
                                              tinyblasGemmAlgo_t algo) {
    if (!check_args(transa, transb, pAlpha, Atype, Btype, pBeta, Ctype,
                    computeType)) {
        return TINYBLAS_STATUS_NOT_SUPPORTED;
    }

    tinyblasGSBE_wrapper<64, 4, 64>(stream, m, n, k, (const half *)A, lda, strideA,
                                     (const half *)B, ldb, strideB, C, Ctype,
                                     ldc, strideC, batchCount);

    return TINYBLAS_STATUS_SUCCESS;
}

const char *tinyblasGetStatusString(tinyblasStatus_t err) {
    switch (err) {
        case TINYBLAS_STATUS_SUCCESS:
            return "TINYBLAS_STATUS_SUCCESS";
        case TINYBLAS_STATUS_NOT_INITIALIZED:
            return "TINYBLAS_STATUS_NOT_INITIALIZED";
        case TINYBLAS_STATUS_ALLOC_FAILED:
            return "TINYBLAS_STATUS_ALLOC_FAILED";
        case TINYBLAS_STATUS_INVALID_VALUE:
            return "TINYBLAS_STATUS_INVALID_VALUE";
        case TINYBLAS_STATUS_ARCH_MISMATCH:
            return "TINYBLAS_STATUS_ARCH_MISMATCH";
        case TINYBLAS_STATUS_MAPPING_ERROR:
            return "TINYBLAS_STATUS_MAPPING_ERROR";
        case TINYBLAS_STATUS_EXECUTION_FAILED:
            return "TINYBLAS_STATUS_EXECUTION_FAILED";
        case TINYBLAS_STATUS_INTERNAL_ERROR:
            return "TINYBLAS_STATUS_INTERNAL_ERROR";
        case TINYBLAS_STATUS_NOT_SUPPORTED:
            return "TINYBLAS_STATUS_NOT_SUPPORTED";
        default:
            return "unknown error";
    }
}
