// https://stackoverflow.com/a/40695640/1653720

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#if !defined(USE_HIP)
#include <hip/hip_runtime_api.h>
#else
#include <hip/hip_runtime_api.h>
#endif

#if defined(USE_HIP)
#define cudaGetDeviceProperties hipGetDeviceProperties
#define cudaGetDeviceCount hipGetDeviceCount
#define cudaError_t hipError_t
#define cudaDeviceProp hipDeviceProp_t
#define cudaSuccess hipSuccess
#define cudaGetErrorString hipGetErrorString
#endif

int main(int argc, char *argv[]) {
  hipDeviceProp_t prop;
  hipError_t status;
  int device_count;
  int device_index = 0;
  if (argc > 1) {
    device_index = atoi(argv[1]);
  }
  status = hipGetDeviceCount(&device_count);
  if (status != hipSuccess) {
    fprintf(stderr, "cudaGetDeviceCount() failed: %s\n", hipGetErrorString(status));
    return -1;
  }
  if (device_index >= device_count) {
    fprintf(stderr, "Specified device index %d exceeds the maximum (the device count on this system is %d)\n", device_index, device_count);
    return -1;
  }
  status = hipGetDeviceProperties(&prop, device_index);
  if (status != hipSuccess) {
    fprintf(stderr, "cudaGetDeviceProperties() for device device_index failed: %s\n", hipGetErrorString(status));
    return -1;
  }

#if !defined(USE_HIP)
  int v = prop.major * 10 + prop.minor;
  printf("%d", v);
#else
  printf("%s", prop.gcnArchName);
#endif
}
